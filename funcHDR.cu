#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime_api.h>
#include ""
#include <math.h>
#define BLOCKSIZE 32

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		system("pause");
		exit(1);
	}
}

__shared__ float sharedMatM[BLOCKSIZE * BLOCKSIZE];
__shared__ float sharedMatm[BLOCKSIZE * BLOCKSIZE];

__global__ void calculateMin(const float* const d_logLuminance,
	float *min_logLum,
	const size_t numRows,
	const size_t numCols){

	//Conseguimos la posici�n del p�xel en la imagen del que se ocupar� el hilo
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	//Calculamos la posici�n del hilo en el bloque
	const int posThreadBlock = threadIdx.x * BLOCKSIZE + threadIdx.y;

	//Si estamos fuera de los l�mites de la imagen, paramos
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	//Almacenamos en memoria compartida el valor correspondiente a cada thread
	sharedMatm[posThreadBlock] = d_logLuminance[thread_1D_pos];

	__syncthreads();
	
	//Ahora iteraremos sobre los elementos de memoria compartida para ir comparando y obtener el elemento menor.
	for (int i = BLOCKSIZE * BLOCKSIZE / 2; i > 0; i /= 2){
		if (posThreadBlock < i){
			if (sharedMatm[posThreadBlock] > sharedMatm[posThreadBlock + i])
				sharedMatm[posThreadBlock] = sharedMatm[posThreadBlock + i];
		}
		__syncthreads();
	}


	if (posThreadBlock == 0){
		if (sharedMatm[0] < min_logLum[blockIdx.x])
			min_logLum[blockIdx.x] = sharedMatm[0];
	}
}

__global__ void calculateMax(const float* const d_logLuminance,
	float *max_logLum,
	const size_t numRows,
	const size_t numCols){

	//Conseguimos la posici�n del p�xel en la imagen del que se ocupar� el hilo
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	//Calculamos la posici�n del hilo en el bloque
	const int posThreadBlock = threadIdx.x * BLOCKSIZE + threadIdx.y;

	//Si estamos fuera de los l�mites de la imagen, paramos
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	//Almacenamos en memoria compartida el valor correspondiente a cada thread
	sharedMatM[posThreadBlock] = d_logLuminance[thread_1D_pos];

	__syncthreads();

	//Ahora iteraremos sobre los elementos de memoria compartida para ir comparando y obtener el elemento mayor.
	for (int i = BLOCKSIZE * BLOCKSIZE / 2; i > 0; i /= 2){
		if (posThreadBlock < i){
			if (sharedMatM[posThreadBlock] < sharedMatM[posThreadBlock + i])
				sharedMatM[posThreadBlock] = sharedMatM[posThreadBlock + i];
		}
		__syncthreads();
	}


	if (posThreadBlock == 0){
		if (sharedMatM[0] > max_logLum[blockIdx.x])
			max_logLum[blockIdx.x] = sharedMatM[0];
	}

}

__global__ void histograma(unsigned char *buffer, long size, unsigned int *histo){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < size){
		bin = (Lum[i] - lumMin) / lumRange * numBins;
	}

}

void calculate_cdf(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  /* TODO
    1) Encontrar el valor m�ximo y m�nimo de luminancia en min_logLum and max_logLum a partir del canal logLuminance 
	2) Obtener el rango a representar
	3) Generar un histograma de todos los valores del canal logLuminance usando la formula 
	bin = (Lum [i] - lumMin) / lumRange * numBins
	4) Realizar un exclusive scan en el histograma para obtener la distribuci�n acumulada (cdf) 
	de los valores de luminancia. Se debe almacenar en el puntero c_cdf
  */    

	//MAXIMO:2.350199

	//TODO: Calcular tama�os de bloque
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	dim3 gridSize((numCols / blockSize.x) + 1, (numRows / blockSize.y) + 1, 1);

	int numBloques = ((numCols * numRows) / (BLOCKSIZE * BLOCKSIZE)) + 2;

	float *myMin, *myMax;

	hipMalloc((float **)&myMin, sizeof(float) * numBloques);
	hipMalloc((float **)&myMax, sizeof(float) * numBloques);

	hipMemset(myMin, min_logLum, sizeof(float) * numBloques);
	hipMemset(myMax, max_logLum, sizeof(float) * numBloques);

	calculateMin << < gridSize, blockSize >> >(d_logLuminance, myMin, numRows, numCols);
	calculateMax << < gridSize, blockSize >> >(d_logLuminance, myMax, numRows, numCols);
	
	//Lanzamos kernels de manera iterativa hasta que solo quede un valor, el valor final.
	for (int i = numBloques; i > 1; i /= BLOCKSIZE * BLOCKSIZE){
		dim3 newGridSize((sqrt(numBloques) / blockSize.x) + 1, (sqrt(numBloques) / blockSize.y) + 1, 1);
		calculateMin << < newGridSize, blockSize >> >(myMin, myMin, sqrt(numBloques) + 1, sqrt(numBloques) + 1);
		calculateMax << < newGridSize, blockSize >> >(myMax, myMax, sqrt(numBloques) + 1, sqrt(numBloques) + 1);
		numBloques /= (BLOCKSIZE * BLOCKSIZE);
	}
	
	//Lanzamos el kernel para la creaci�n de histogramas.



	hipMemcpy(&min_logLum, myMin, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&max_logLum, myMax, sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());
}