#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime_api.h>
#include ""

#define BLOCKSIZE 32

__shared__ float sharedMatM[BLOCKSIZE * BLOCKSIZE];
__shared__ float sharedMatm[BLOCKSIZE * BLOCKSIZE];

__global__ void calculateMaxMin(const float* const d_logLuminance,
	float &min_logLum,
	float &max_logLum,
	const size_t numRows,
	const size_t numCols){

	//Conseguimos la posici�n del p�xel en la imagen del que se ocupar� el hilo
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	//Calculamos la posici�n del hilo en el bloque
	const int posThreadBlock = threadIdx.x * BLOCKSIZE + threadIdx.y;

	//Si estamos fuera de los l�mites de la imagen, paramos
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	//Almacenamos en memoria compartida el valor correspondiente a cada thread
	sharedMatM[posThreadBlock] = d_logLuminance[thread_1D_pos];
	sharedMatm[posThreadBlock] = d_logLuminance[thread_1D_pos];

	__syncthreads();

	//Ahora iteraremos sobre los elementos de memoria compartida para ir comparando y obtener el elemento menor y el mayor correspondientemente.
	for (int i = BLOCKSIZE / 2; i > 0; i /= 2){
		if (posThreadBlock < i){
			if (sharedMatm[posThreadBlock] > sharedMatm[posThreadBlock + i])
				sharedMatm[posThreadBlock] = sharedMatm[posThreadBlock + i];

			if (sharedMatM[posThreadBlock] < sharedMatM[posThreadBlock + i])
				sharedMatM[posThreadBlock] = sharedMatM[posThreadBlock + i];
		}
		__syncthreads();
	}

	if (posThreadBlock == 0){
		min_logLum = sharedMatm[0];
		max_logLum = sharedMatM[0];
		printf("Guardo el valor %f\n", d_logLuminance[thread_1D_pos]);

	}

}


void calculate_cdf(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  /* TODO
    1) Encontrar el valor m�ximo y m�nimo de luminancia en min_logLum and max_logLum a partir del canal logLuminance 
	2) Obtener el rango a representar
	3) Generar un histograma de todos los valores del canal logLuminance usando la formula 
	bin = (Lum [i] - lumMin) / lumRange * numBins
	4) Realizar un exclusive scan en el histograma para obtener la distribuci�n acumulada (cdf) 
	de los valores de luminancia. Se debe almacenar en el puntero c_cdf
  */    

	//TODO: Calcular tama�os de bloque
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	const dim3 gridSize((numCols / blockSize.x) + 1, (numRows / blockSize.y) + 1, 1);

	//TODO: Lanzar kernel para separar imagenes RGBA en diferentes colores
	calculateMaxMin << < gridSize, blockSize >> >(d_logLuminance, min_logLum, max_logLum, numRows, numCols);

	printf("Minimo = %f\nMaximo = %f\n", min_logLum, max_logLum);

	hipDeviceSynchronize(); 
	//checkCudaErrors(hipGetLastError());
}
