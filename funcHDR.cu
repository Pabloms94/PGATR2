#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime_api.h>
#include ""
#include <math.h>

#define BLOCKSIZE 32

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		system("pause");
		exit(1);
	}
}

__shared__ float sharedMatM[BLOCKSIZE * BLOCKSIZE];
__shared__ float sharedMatm[BLOCKSIZE * BLOCKSIZE];

__global__ void calculateMin(const float* const d_logLuminance,
	float *min_logLum,
	const size_t numRows,
	const size_t numCols){

	//Conseguimos la posici�n del p�xel en la imagen del que se ocupar� el hilo
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	//Calculamos la posici�n del hilo en el bloque
	const int posThreadBlock = threadIdx.x * BLOCKSIZE + threadIdx.y;

	//Si estamos fuera de los l�mites de la imagen, paramos
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	//Almacenamos en memoria compartida el valor correspondiente a cada thread
	sharedMatm[posThreadBlock] = d_logLuminance[thread_1D_pos];

	__syncthreads();
	
	//Ahora iteraremos sobre los elementos de memoria compartida para ir comparando y obtener el elemento menor.
	for (int i = BLOCKSIZE * BLOCKSIZE / 2; i > 0; i /= 2){
		if (posThreadBlock < i){
			if (sharedMatm[posThreadBlock] > sharedMatm[posThreadBlock + i])
				sharedMatm[posThreadBlock] = sharedMatm[posThreadBlock + i];
		}
		__syncthreads();
	}


	if (posThreadBlock == 0){
		if (sharedMatm[0] < min_logLum[blockIdx.x])
			min_logLum[blockIdx.x] = sharedMatm[0];
	}
}

__global__ void calculateMax(const float* const d_logLuminance,
	float *max_logLum,
	const size_t numRows,
	const size_t numCols){

	//Conseguimos la posici�n del p�xel en la imagen del que se ocupar� el hilo
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	//Calculamos la posici�n del hilo en el bloque
	const int posThreadBlock = threadIdx.x * BLOCKSIZE + threadIdx.y;

	//Si estamos fuera de los l�mites de la imagen, paramos
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	//Almacenamos en memoria compartida el valor correspondiente a cada thread
	sharedMatM[posThreadBlock] = d_logLuminance[thread_1D_pos];

	__syncthreads();

	//Ahora iteraremos sobre los elementos de memoria compartida para ir comparando y obtener el elemento mayor.
	for (int i = BLOCKSIZE * BLOCKSIZE / 2; i > 0; i /= 2){
		if (posThreadBlock < i){
			if (sharedMatM[posThreadBlock] < sharedMatM[posThreadBlock + i])
				sharedMatM[posThreadBlock] = sharedMatM[posThreadBlock + i];
		}
		__syncthreads();
	}


	if (posThreadBlock == 0){
		if (sharedMatM[0] > max_logLum[blockIdx.x])
			max_logLum[blockIdx.x] = sharedMatM[0];
	}

}

__global__ void histograma(const float* const d_logLuminance, 
	float min_logLum,
	float max_logLum, 
	const size_t numRows,
	const size_t numCols,
	const size_t numBins, 
	unsigned int *histo){

	//Conseguimos la posici�n del p�xel en la imagen del que se ocupar� el hilo
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	//Calculamos la posici�n del hilo en el bloque
	const int posThreadBlock = threadIdx.x * BLOCKSIZE + threadIdx.y;

	//Si estamos fuera de los l�mites de la imagen, paramos
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;
	
	/*if (posThreadBlock == 0)
		printf("MIN %f Y MAX %f\n", min_logLum, max_logLum);*/

	int bin = (int) ((d_logLuminance[posThreadBlock] - min_logLum) / (fabs(max_logLum - min_logLum)) * numBins);
	atomicAdd(&histo[bin], 1);
	//printf("Valor %d es %d\n", bin, histo[bin]);
}

__global__ void exclusiveScan(unsigned int *histo, const size_t numBins){

	__shared__ int tempArray[BLOCKSIZE * BLOCKSIZE];

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int threadId = threadIdx.x;
	int offset = 1, temp;
	int ai = threadId;
	int bi = threadId + numBins / 2;
	
	tempArray[ai] = histo[id];
	tempArray[bi] = histo[id + numBins / 2];

	//if (id>0 && id<100)
	//	printf("ID:%d Histograma: %u\n", (id + numBins / 2), histo[id + numBins / 2]);
	//printf("ID Thread2: %d\n", id);
	for (int i = numBins >> 1; i > 0; i >>= 1){
		__syncthreads();
		if (threadId < i){
			ai = offset * (2 * threadId + 1) - 1;
			bi = offset * (2 * threadId + 2) - 1;
			tempArray[bi] += tempArray[ai];
			printf("ID Thread2: %d\n", tempArray[bi]);
		}
		offset <<= 1;
	}
	

	if (threadId == 0){
		tempArray[numBins - 1] = 0;
	}

	for (int i = 1; i < numBins; i <<= 1){
		offset >>= 1;
		__syncthreads();
		if (threadId < i){
			ai = offset * (2 * threadId + 1) - 1;
			bi = offset * (2 * threadId + 2) - 1;
			temp = tempArray[ai];
			tempArray[ai] = tempArray[bi];
			tempArray[bi] += temp;
		}
	}
	
	//if (id>0 && id<100)
		//printf("ID REAL: %u ID:%d Histograma: %u\n",id, (id + numBins / 2), histo[id + numBins / 2]);

	__syncthreads();

	//histo[id] = tempArray[threadId];
	//histo[id + numBins / 2] = tempArray[threadId + numBins / 2];//printf("ID Thread4: %d\n", id);
	

}

void calculate_cdf(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  /* TODO
    1) Encontrar el valor m�ximo y m�nimo de luminancia en min_logLum and max_logLum a partir del canal logLuminance 
	2) Obtener el rango a representar
	3) Generar un histograma de todos los valores del canal logLuminance usando la formula 
	bin = (Lum [i] - lumMin) / lumRange * numBins
	4) Realizar un exclusive scan en el histograma para obtener la distribuci�n acumulada (cdf) 
	de los valores de luminancia. Se debe almacenar en el puntero c_cdf
  */    

	//MAXIMO:2.350199

	//TODO: Calcular tama�os de bloque
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	dim3 gridSize((numCols / blockSize.x) + 1, (numRows / blockSize.y) + 1, 1);

	int numBloques = ((numCols * numRows) / (BLOCKSIZE * BLOCKSIZE)) + 2;

	float *myMin, *myMax;

	hipMalloc((float **)&myMin, sizeof(float) * numBloques);
	hipMalloc((float **)&myMax, sizeof(float) * numBloques);

	hipMemset(myMin, min_logLum, sizeof(float) * numBloques);
	hipMemset(myMax, max_logLum, sizeof(float) * numBloques);

	calculateMin << < gridSize, blockSize >> >(d_logLuminance, myMin, numRows, numCols);
	calculateMax << < gridSize, blockSize >> >(d_logLuminance, myMax, numRows, numCols);
	
	//Lanzamos kernels de manera iterativa hasta que solo quede un valor, el valor final.
	for (int i = numBloques; i > 1; i /= BLOCKSIZE * BLOCKSIZE){
		dim3 newGridSize((sqrt(numBloques) / blockSize.x) + 1, (sqrt(numBloques) / blockSize.y) + 1, 1);
		calculateMin << < newGridSize, blockSize >> >(myMin, myMin, sqrt(numBloques) + 1, sqrt(numBloques) + 1);
		calculateMax << < newGridSize, blockSize >> >(myMax, myMax, sqrt(numBloques) + 1, sqrt(numBloques) + 1);
		numBloques /= (BLOCKSIZE * BLOCKSIZE);
	}

	hipMemcpy(&min_logLum, myMin, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&max_logLum, myMax, sizeof(float), hipMemcpyDeviceToHost);

	//printf("MIN %f Y MAX %f\n", min_logLum, max_logLum);

	unsigned int *myHisto;
	unsigned int *hostHisto = new unsigned int[numBins];
	
	hipMalloc((int **)&myHisto, sizeof(unsigned int) * numBins);
	//hipMalloc((int **)&hostHisto, sizeof(unsigned int) * numBins);
	hipMemset(myHisto, 0, sizeof(unsigned int) * numBins);

	//Lanzamos el kernel para la creaci�n de histogramas.
	histograma << < gridSize, blockSize >> >(d_logLuminance, min_logLum, max_logLum, numRows, numCols, numBins, myHisto);

	/*hipMemcpy(&min_logLum, myMin, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&max_logLum, myMax, sizeof(float), hipMemcpyDeviceToHost);*/

	//hipMalloc((unsigned int **)&hostHisto, sizeof(unsigned int) * numBins);
	hipMemcpy(hostHisto, myHisto, sizeof(unsigned int) * numBins, hipMemcpyDeviceToHost);

	//for (unsigned int i = 0; i < numBins; i++)
	//{
	//	printf("id: %d  value: %u\n", i, hostHisto[i]);
	//}

	exclusiveScan << < gridSize, blockSize >> >(myHisto, numBins);

	//hipMemcpy(d_cdf, myHisto, sizeof(unsigned int) * numBins, hipMemcpyDeviceToHost);

	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());
}